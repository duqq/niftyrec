#include "hip/hip_runtime.h"

#ifndef _TTBACKPROJECTRAY_CU_
#define _TTBACKPROJECTRAY_CU_

#include <_tt_backproject_ray_gpu.h>

//########################################
//#### Test CPU backprojection ###########
//########################################

struct Ray_cpu {
	float3 o;	// origin
	float3 d;	// direction
};
int intersectBox_cpu(Ray_cpu r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = {1.0/r.d.x,1.0/r.d.y,1.0/r.d.z};
    float3 tbot = {invR.x*(boxmin.x - r.o.x), invR.y*(boxmin.y - r.o.y), invR.z*(boxmin.z - r.o.z)};
    float3 ttop = {invR.x*(boxmax.x - r.o.x), invR.y*(boxmax.y - r.o.y), invR.z*(boxmax.z - r.o.z)};

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}
float3 mul_cpu(float *M, float3 v)
{
    float3 r;
    float3 t = {M[0], M[1], M[2]};
    r.x = dot(v, t);
    t.x=M[4]; t.y=M[5]; t.z=M[6];
    r.y = dot(v, t);
    t.x=M[8]; t.y=M[9]; t.z=M[10];
    r.z = dot(v, t);
    return r;
}
float4 mul_cpu(float *M, float4 v)
{
    float4 r;
    float4 t = {M[0], M[1], M[2], M[3]};
    r.x = dot(v, t);
    t.x=M[4]; t.y=M[5]; t.z=M[6]; t.w=M[7];
    r.y = dot(v, t);
    t.x=M[8]; t.y=M[9]; t.z=M[10]; t.w=M[11];
    r.z = dot(v, t);
    r.w = 1.0f;
    return r;}

extern "C" int tt_backproject_cpu(float *out_backprojection, float *current_projection, float *invViewMatrix, uint2 detectorPixels, float3 sourcePosition, uint3 volumeVoxels, float3 volumeSize, float t_step, int interpolation)
{   
    const int    maxSteps = 100000; 
    const float3 boxMin = {0.0f, 0.0f, 0.0f};
    const float3 boxMax = {volumeSize.x, volumeSize.y, volumeSize.z};
    float u,v;
    float4 temp4; 
    float3 temp3; 
    int hits=0;

    for (int x=1; x<detectorPixels.x; x++)
    {
        for (int y=1; y<detectorPixels.y; y++)
        {
            u = (x / (float) detectorPixels.x);
            v = (y / (float) detectorPixels.y);

            Ray_cpu eyeRay;
            eyeRay.o = sourcePosition;
            //transform and normalize direction vector
            temp4.x=u; temp4.y=v; temp4.z=0.0f; temp4.w=1.0f;
            temp4 = mul_cpu(invViewMatrix, temp4);
            temp3.x = temp4.x; temp3.y = temp4.y; temp3.z = temp4.z; 
            eyeRay.d = normalize(temp3-eyeRay.o); 
            // find intersection with box
            float tnear, tfar;
            int hit = intersectBox_cpu(eyeRay, boxMin, boxMax, &tnear, &tfar);
//            if (!hit) fprintf(stderr,"\n -> %d %d - %f %f %f - %f %f %f - %f %f %f - %f %f %f",x,y,boxMin.x, boxMin.y, boxMin.z, boxMax.x, boxMax.y, boxMax.z, eyeRay.o.x,eyeRay.o.y,eyeRay.o.z, eyeRay.d.x,eyeRay.d.y,eyeRay.d.z );
            if (tnear < 0.0f) tnear = 0.0f;
            if (hit)
            {
                hits++;

                // march along ray from front to back, accumulating
                float  t = tnear;
                float3 pos = eyeRay.o + eyeRay.d*tnear;
                float3 step = eyeRay.d*t_step;
                float  bkpr = current_projection[y*detectorPixels.x+x];

                for(int i=0; i<maxSteps; i++)
                {
                    if (interpolation == INTERP_NEAREST)
                    {
                        int px, py, pz;
                        px = (int) pos.x; py = (int) pos.y; pz = (int) pos.z; 
                        int index = pz*volumeVoxels.y*volumeVoxels.x+py*volumeVoxels.x+px;
                        out_backprojection[index] += bkpr;
                    }
                    else if (interpolation == INTERP_TRILINEAR)
                    {
                        int3 p000, p001, p010, p011, p100, p101, p110, p111;
                        p000.x = (int)floor(pos.x); p000.y = (int)floor(pos.y); p000.z = (int)floor(pos.z); 
                        p001 = p000; p001.x+=1;
                        p010 = p000; p010.y+=1;
                        p011 = p010; p011.x+=1;
                        p100 = p000; p100.z+=1;
                        p101 = p001; p101.z+=1;
                        p110 = p100; p110.y+=1;
                        p111 = p110; p111.x+=1; 
                        float3 d;
                        d.x = pos.x-p000.x;
                        d.y = pos.y-p000.y;
                        d.z = pos.z-p000.z; 
                        float w000, w001, w010, w011, w100, w101, w110, w111;                      
                        w000 = (1-d.z)*(1-d.y)*(1-d.x)*bkpr;
                        w001 = (1-d.z)*(1-d.y)*( d.x )*bkpr;
                        w010 = (1-d.z)*( d.y )*(1-d.x)*bkpr;
                        w011 = (1-d.z)*( d.y )*( d.x )*bkpr;
                        w100 = ( d.z )*(1-d.y)*(1-d.x)*bkpr;
                        w101 = ( d.z )*(1-d.y)*( d.x )*bkpr;
                        w110 = ( d.z )*( d.y )*(1-d.x)*bkpr;
                        w111 = ( d.z )*( d.y )*( d.x )*bkpr;
                        out_backprojection[p000.z*volumeVoxels.y*volumeVoxels.x+p000.y*volumeVoxels.x+p000.x] += w000;
                        out_backprojection[p001.z*volumeVoxels.y*volumeVoxels.x+p001.y*volumeVoxels.x+p001.x] += w001;
                        out_backprojection[p010.z*volumeVoxels.y*volumeVoxels.x+p010.y*volumeVoxels.x+p010.x] += w010;
                        out_backprojection[p011.z*volumeVoxels.y*volumeVoxels.x+p011.y*volumeVoxels.x+p011.x] += w011;
                        out_backprojection[p100.z*volumeVoxels.y*volumeVoxels.x+p100.y*volumeVoxels.x+p100.x] += w100;
                        out_backprojection[p101.z*volumeVoxels.y*volumeVoxels.x+p101.y*volumeVoxels.x+p101.x] += w101;
                        out_backprojection[p110.z*volumeVoxels.y*volumeVoxels.x+p110.y*volumeVoxels.x+p110.x] += w110;
                        out_backprojection[p111.z*volumeVoxels.y*volumeVoxels.x+p111.y*volumeVoxels.x+p111.x] += w111;
                    }

                    t += t_step;
                    if (t > tfar) break;
                    pos += step;
                }
            }
        }
    }
    fprintf(stderr,"\nHits: %d",hits);
    return 0;
}
//########################################
//########################################
//########################################


int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

extern "C" int set_inViewMatrix(float *invViewMatrix, float_2 detector_scale, float_3 detector_transl, float_3 detector_rotat)
{
    memset((void*)invViewMatrix,0,12*sizeof(float));
    //rotate
    mat_44 *rotation = (mat_44 *)calloc(1,sizeof(mat_44));
    create_rotation_matrix44(rotation, detector_rotat.x,detector_rotat.y,detector_rotat.z,0,0,0);
    //scale
    mat_44 *scale = (mat_44 *)calloc(1,sizeof(mat_44));
    scale->m[0][0] =detector_scale.x;
    scale->m[1][1] =detector_scale.y;
    scale->m[2][2] =1;
    //transform
    mat_44 *m = (mat_44 *)calloc(1,sizeof(mat_44));
    *m = reg_mat_44_mul(rotation,scale);
    invViewMatrix[0]=m->m[0][0]; invViewMatrix[1]=m->m[0][1]; invViewMatrix[2] =m->m[0][2]; 
    invViewMatrix[4]=m->m[1][0]; invViewMatrix[5]=m->m[1][1]; invViewMatrix[6] =m->m[1][2]; 
    invViewMatrix[8]=m->m[2][0]; invViewMatrix[9]=m->m[2][1]; invViewMatrix[10]=m->m[2][2];
    //translate
    invViewMatrix[3] =detector_transl.x;
    invViewMatrix[7] =detector_transl.y;
    invViewMatrix[11]=detector_transl.z; 
    //cleanup
    free(rotation);
    free(scale);
    free(m);
    return 0;
}


#endif

