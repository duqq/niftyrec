#include "hip/hip_runtime.h"
#include "_et_line_integral_gpu.h"
#include "_et_line_integral_gpu_kernels.cu"

#define BLOCK 256

void et_line_integral_gpu(float **d_activity, float **d_sinogram, int cam, nifti_image *img)
{
	int3 imageSize = make_int3(img->dim[1],img->dim[2],img->dim[3]);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageSize),&imageSize,sizeof(int3)));
	
	const unsigned int Grid = (unsigned int)ceil(img->dim[1]*img->dim[2]/(float)BLOCK);
	dim3 B1(BLOCK,1,1);
	dim3 G1(Grid,1,1);
	
	float *currentCamPointer = (*d_sinogram) + cam * img->dim[1] * img->dim[2] ;
	
	et_line_integral_gpu_kernel <<<G1,B1>>> (*d_activity, currentCamPointer);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
}


