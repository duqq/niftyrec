
#include "_et_convolveFFT2D_gpu_kernels.cu"
#include "_et_common.h"

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
int calculateFFTsize(int dataSize){
    int hiBit;
    unsigned int lowPOT, hiPOT;

    dataSize = iAlignUp(dataSize, 16);

    for(hiBit = 31; hiBit >= 0; hiBit--)
        if(dataSize & (1U << hiBit)) break;

    lowPOT = 1U << hiBit;
    if(lowPOT == dataSize)
        return dataSize;

    hiPOT = 1U << (hiBit + 1);
    if(hiPOT <= 1024)
        return hiPOT;
    else 
        return iAlignUp(dataSize, 512);
}



////////////////////////////////////////////////////////////////////////////////
// 2D Convolution
////////////////////////////////////////////////////////////////////////////////

int et_convolveFFT2D_gpu(float **d_data, int *data_size, float **d_kernel, int *kernel_size, float **d_result)
{
    int status = 1;
    const int dataH = data_size[0];
    const int dataW = data_size[1];
    const int kernelH = kernel_size[0];
    const int kernelW = kernel_size[1];

    const int kernelX = (kernelH-1)/2;
    const int kernelY = (kernelW-1)/2;

    const int n_slices = data_size[2];
    const int data_slice_size = dataH * dataW;
    const int kernel_slice_size = kernelH * kernelW;

    float *d_PaddedData, *d_PaddedKernel, *d_Data, *d_Kernel, *d_Result;
    fComplex *d_DataSpectrum, *d_KernelSpectrum;
    hipfftHandle fftPlanFwd, fftPlanInv;

    //Derive FFT size from data and kernel dimensions
    const int fftW = calculateFFTsize(dataW + kernelW - 1);
    const int fftH = calculateFFTsize(dataH + kernelH - 1);

    //Allocate memory for zero-padded image and kernel and for their transforms
    fprintf_verbose("Allocating memory...\n");
    cutilSafeCall( hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)) );
    cutilSafeCall( hipMalloc((void **)&d_PaddedData,   fftH * fftW * sizeof(float)) );

    cutilSafeCall( hipMalloc((void **)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)) );
    cutilSafeCall( hipMalloc((void **)&d_DataSpectrum,   fftH * (fftW / 2 + 1) * sizeof(fComplex)) );

    //Create cuFFT plan
    fprintf_verbose("Creating FFT plan for %i x %i...\n", fftH, fftW);
    cufftSafeCall( hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C) );
    cufftSafeCall( hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R) );

    //Convolve slices one by one
    for (int slice=0; slice<n_slices; slice++)
        {
        //Determine slice pointer
        d_Data = (*d_data) + slice * data_slice_size; 
        d_Kernel = (*d_kernel) + slice * kernel_slice_size;
        d_Result = (*d_result) + slice * data_slice_size;

        //Zero pad
        fprintf_verbose("Padding convolution kernel and input data...\n");
        cutilSafeCall( hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)) );
        cutilSafeCall( hipMemset(d_PaddedData,   0, fftH * fftW * sizeof(float)) );
        padKernel(d_PaddedKernel,d_Kernel,fftH,fftW,kernelH,kernelW,kernelY,kernelX);
cutilSafeCall( hipDeviceSynchronize() );
	if (!d_PaddedData || !d_PaddedKernel) fprintf_verbose("NULL arguments!\n");
        fprintf_verbose( "%d %d %d %d %d %d %d %d %d %d\n", d_PaddedData,d_Data,fftH,fftW,dataH,dataW,kernelH,kernelW,kernelY,kernelX);
        padDataClampToBorder(d_PaddedData,d_Data,fftH,fftW,dataH,dataW,kernelH,kernelW,kernelY,kernelX);
cutilSafeCall( hipDeviceSynchronize() );
        //Convolve
        fprintf_verbose("Transforming convolution kernel...\n");
        cufftSafeCall( hipfftExecR2C(fftPlanFwd, d_PaddedKernel, (hipfftComplex *)d_KernelSpectrum) );

        fprintf_verbose("Running GPU FFT convolution...\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cufftSafeCall( hipfftExecR2C(fftPlanFwd, d_PaddedData, (hipfftComplex *)d_DataSpectrum) );
        modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW);
        cufftSafeCall( hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_DataSpectrum, d_PaddedData) );
        cutilSafeCall( hipDeviceSynchronize() );
      

        //Crop result
        fprintf_verbose("Cropping result image...\n");
        //cutilSafeCall( hipMemset(d_Result, 11, dataH * dataW * sizeof(float)) ); //FIXME do the real thing
        crop_image(d_Result,d_PaddedData,fftH,fftW,dataH,dataW,kernelH,kernelW);
        }

    //Destroy cuFFT plan and free memory
    fprintf_verbose("Shutting down...\n");
    cufftSafeCall( hipfftDestroy(fftPlanInv) );
    cufftSafeCall( hipfftDestroy(fftPlanFwd) );
    cutilSafeCall( hipFree(d_DataSpectrum)   );
    cutilSafeCall( hipFree(d_KernelSpectrum) );
    cutilSafeCall( hipFree(d_PaddedData)   );
    cutilSafeCall( hipFree(d_PaddedKernel) );

    status = 0;
    return status;
}



